#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <cub/block/block_discontinuity.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_reduce.cuh>
#include <cub/block/block_store.cuh>
#include <hipcub/hipcub.hpp>
#include <cub/warp/warp_reduce.cuh>
#include <torch/extension.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <limits.h>
#include <mma.h>
#include <stdio.h>
#include <vector>
#define CDIV(x, y) (((x) + (y)-1) / (y))

using namespace cooperative_groups;
namespace cg = cooperative_groups;

void CUDA_CHECK_RETURN_(hipError_t cudaStatus) {
    if (cudaStatus != hipSuccess) {
        printf("CUDA Failure: %s\n", hipGetErrorString(cudaStatus));
        // exit(EXIT_FAILURE); // so many segfaults before being able to print out actual crap because of this stupidity
    }
}

__device__ float dequantize_fp4_tree(unsigned char val, float absmax) {
    float sign = (val & 0b1000) == 8 ? -1.0f : 1.0f;
    if ((val & 0b0100) == 4) // 0
        if ((val & 0b0010) == 2) // 01
            if ((val & 0b0001) == 1) // 111
                return 0.25000000f * absmax * sign; // 1111
            else
                return 0.16666667f * absmax * sign; // 1110
        else if ((val & 0b0001) == 1) // 110
            return 0.50000000f * absmax * sign; // 1101
        else
            return 0.33333333f * absmax * sign; // 1100
    else if ((val & 0b0010) == 2) // 10
        if ((val & 0b0001) == 1) // 101
            return 1.00000000f * absmax * sign; // 1011
        else
            return 0.66666667f * absmax * sign; // 1010
    else if ((val & 0b0001) == 1) // 100
        return 5.208333333e-03f * absmax * sign; // 1001
    else
        return 0.00000000f * absmax * sign; // 1000
}

template <typename T> __device__ __forceinline__ T convert_to_ty(float val);
template <> __device__ __forceinline__ hip_bfloat16 convert_to_ty(float val) {
    return __float2bfloat16(val);
}
template <> __device__ __forceinline__ nv_half convert_to_ty(float val) {
    return __float2half(val);
}
template <> __device__ __forceinline__ float convert_to_ty(float val) {
    return val;
}

template <typename T, int TILE_SIZE, int THREADS, int NUM_PER_TH>
__global__ void dequantize_blockwise_kernel_fp4(unsigned char *A, float *absmax, T *out, const int blocksize, const int n) {
    const int n_load = (gridDim.x * TILE_SIZE);
    int valid_items_load = 0;
    int valid_items_store = 0;
    const int base_idx = (blockIdx.x * TILE_SIZE);
    T vals[NUM_PER_TH * 2];
    unsigned char qvals[NUM_PER_TH];
    float local_abs_max;

    valid_items_load = 0;
    valid_items_store = 0;
    local_abs_max = -FLT_MAX;
    typedef hipcub::BlockLoad<unsigned char, THREADS, NUM_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;
    typedef hipcub::BlockStore<T, THREADS, NUM_PER_TH * 2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ typename LoadChar::TempStorage loadchar;
    __shared__ typename StoreT::TempStorage storet;
    for (unsigned int i = base_idx; i < n_load; i += gridDim.x * TILE_SIZE) {
        valid_items_load = (n + 1) / 2 - i > TILE_SIZE ? TILE_SIZE : (n + 1) / 2 - i;
        valid_items_store = n - i * 2 > TILE_SIZE * 2 ? TILE_SIZE * 2 : n - i * 2;
        local_abs_max = __ldg(&absmax[(i + threadIdx.x * NUM_PER_TH) / (blocksize)]);
        __syncthreads();

        LoadChar(loadchar).Load(&(A[i]), qvals, valid_items_load, 128);
#pragma unroll NUM_PER_TH
        for (int j = 0; j < NUM_PER_TH; j++) {

            vals[j * 2] = convert_to_ty<T>(dequantize_fp4_tree(qvals[j] >> 4, local_abs_max));
            vals[j * 2 + 1] = convert_to_ty<T>(dequantize_fp4_tree(qvals[j] & 0x0F, local_abs_max));
        }
        __syncthreads();
        StoreT(storet).Store(&(out[i * 2]), vals, valid_items_store);
    }
}

template <typename T, int TILE_SIZE, int THREADS, int NUM_PER_TH>
__global__ void
dequantize_blockwise_codebook_kernel_fp4(unsigned char *A, float *absmax, T *out, float *code, const int blocksize, const int n) {
    const int warp_idx = threadIdx.x / 32;
    const int warp_lane = threadIdx.x % 32;

    const int n_load = (gridDim.x * TILE_SIZE);
    int valid_items_load = 0;
    int valid_items_store = 0;
    const int base_idx = (blockIdx.x * TILE_SIZE);
    T vals[NUM_PER_TH * 2];
    unsigned char qvals[NUM_PER_TH];
    float local_abs_max = -FLT_MAX;
    __shared__ float local_code[16];

    typedef hipcub::BlockLoad<unsigned char, THREADS, NUM_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;
    typedef hipcub::BlockStore<T, THREADS, NUM_PER_TH * 2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    if (warp_lane < 16 && warp_idx == 0) local_code[warp_lane] = code[warp_lane];
    __threadfence_block();

    __shared__ typename LoadChar::TempStorage loadchar;
    __shared__ typename StoreT::TempStorage storet;
    for (unsigned int i = base_idx; i < n_load; i += gridDim.x * TILE_SIZE) {
        valid_items_load = (n + 1) / 2 - i > TILE_SIZE ? TILE_SIZE : (n + 1) / 2 - i;
        valid_items_store = n - i * 2 > TILE_SIZE * 2 ? TILE_SIZE * 2 : n - i * 2;
        local_abs_max = __ldg(&absmax[(i + threadIdx.x * NUM_PER_TH) / (blocksize)]);
        __syncthreads();

        LoadChar(loadchar).Load(&(A[i]), qvals, valid_items_load, 128);
#pragma unroll NUM_PER_TH
        for (int j = 0; j < NUM_PER_TH; j++) {

            vals[j * 2] = T(code[qvals[j] >> 4] * local_abs_max);
            vals[j * 2 + 1] = T(code[qvals[j] & 0x0F] * local_abs_max);
        }
        __syncthreads();
        StoreT(storet).Store(&(out[i * 2]), vals, valid_items_store);
    }
}

template <typename T>
void launch_dequantize_blockwise_kernel_fp4(torch::Tensor A, torch::Tensor absmax, torch::Tensor out, int blocksize, int n) {
    const int blocks = CDIV(n, 1024);
    dequantize_blockwise_kernel_fp4<T, 512, 64, 8><<<blocks, 64>>>(
        (unsigned char *)A.data_ptr(), (float *)absmax.data_ptr(), (T *)out.mutable_data_ptr(), (const int)(blocksize / 2), (const int)n
    );
    CUDA_CHECK_RETURN_(hipGetLastError());
}

void dequantize_blockwise_fp4(torch::Tensor A, torch::Tensor absmax, int M, int N, int blocksize, int n, torch::Tensor out) {
    TORCH_CHECK(A.dtype() == torch::kUInt8, "A must be uint8");
    TORCH_CHECK(absmax.dtype() == torch::kFloat32, "absmax must be float32");
    TORCH_CHECK(A.is_cuda(), "A must be cuda");
    TORCH_CHECK(absmax.is_cuda(), "absmax must be cuda");
    TORCH_CHECK(out.is_cuda(), "out must be cuda");
    switch (out.scalar_type()) {
        case torch::kFloat16: {
            launch_dequantize_blockwise_kernel_fp4<nv_half>(A, absmax, out, blocksize, n);
            break;
        }
        case torch::kFloat32: {
            launch_dequantize_blockwise_kernel_fp4<float>(A, absmax, out, blocksize, n);
            break;
        }
        case torch::kBFloat16: {
            launch_dequantize_blockwise_kernel_fp4<hip_bfloat16>(A, absmax, out, blocksize, n);
            break;
        }
        default: {
            std::cout << "NO APPLICABLE DEQUANT DTYPE!" << std::endl;
        }
    }
}

torch::Tensor dequantize_blockwise_codebook_fp4(
    torch::Tensor A, torch::Tensor absmax, torch::Tensor codebook, int M, int N, int blocksize, int n, torch::ScalarType dtype
) {
    TORCH_CHECK(A.dtype() == torch::kUInt8, "A must be uint8");
    TORCH_CHECK(absmax.dtype() == torch::kFloat32, "absmax must be float32");
    TORCH_CHECK(A.is_cuda(), "A must be cuda");
    TORCH_CHECK(absmax.is_cuda(), "absmax must be cuda");
    TORCH_CHECK_TYPE(codebook.dtype() == torch::kFloat32, "codebook must be float32");
    torch::Tensor out = torch::empty({M, N}, torch::dtype(dtype).device(A.device()));
    const int blocks = CDIV(n, 1024);
    switch (dtype) {
        case torch::kFloat32: {
            dequantize_blockwise_codebook_kernel_fp4<float, 512, 64, 8><<<blocks, 64>>>(
                (unsigned char *)A.data_ptr(),
                (float *)absmax.data_ptr(),
                (float *)out.mutable_data_ptr(),
                (float *)codebook.data_ptr(),
                (const int)(blocksize / 2),
                (const int)n
            );
            break;
        }
        case torch::kFloat16: {
            dequantize_blockwise_codebook_kernel_fp4<nv_half, 512, 64, 8><<<blocks, 64>>>(
                (unsigned char *)A.data_ptr(),
                (float *)absmax.data_ptr(),
                (nv_half *)out.mutable_data_ptr(),
                (float *)codebook.data_ptr(),
                (const int)(blocksize / 2),
                (const int)n
            );
            break;
        }
        case torch::kBFloat16: {
            dequantize_blockwise_codebook_kernel_fp4<hip_bfloat16, 512, 64, 8><<<blocks, 64>>>(
                (unsigned char *)A.data_ptr(),
                (float *)absmax.data_ptr(),
                (hip_bfloat16 *)out.mutable_data_ptr(),
                (float *)codebook.data_ptr(),
                (const int)(blocksize / 2),
                (const int)n
            );
            break;
        }
        default: {
            std::cout << "NO APPLICABLE DTYPE!" << std::endl;
        }
    }
    return out;
}